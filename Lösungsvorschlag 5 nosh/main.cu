#include <iostream>
#include <vector>
#include <string>

#include <opencv2/opencv.hpp>
using namespace cv;

#ifdef WIN32
#include <time.h>
#endif

#include "sobel.cu"// important to include .cu file, not header file

void detectEdges (const std::vector<std::string>& images);
std::string resultName (const std::string& fname, const std::string& resultTxt);

int main (int argc, char** argv)
{
    std::vector<std::string> fname;
    enum {FULL, EDGES, CIRCLES, LINES, CALIBRATE, TRACK} mode = FULL;

    for (int i=1; i<argc; i++)
        if (std::string(argv[i])==std::string("-edges")) mode = EDGES;
        else if (argv[i][0]!='-') fname.push_back (std::string(argv[i]));
        else {
            std::cerr << "Unknown option " << argv[i] << std::endl;
            return 2;
        }
    if (fname.empty()) {
        std::cout << "No image specified." << std::endl;
        return 2;
    }
    if (mode==EDGES) {
        hipFree(0);//initialize CUDA
        detectEdges (fname);
    }
    else {
        std::cout << "Illegal command " << std::endl;
        return 1;
    }
    return 0;
}

void detectEdges (const std::vector<std::string>& images)
{
    if (images.size()==0) return;
    for(int imageCtr=0; imageCtr<(int) images.size(); imageCtr++) {
        std::string fname = images[imageCtr];
        Mat_<uchar> srcImg = imread (fname, 0); // load image as greyscale
        if (srcImg.empty()) {
            std::cerr << "Could not find " << fname << std::endl;
            continue;
        }
        Mat_<uchar> sobelImg1 = Mat::zeros(srcImg.rows, srcImg.cols, CV_8UC1);

        int64 time, time2;
        time = cvGetTickCount();
        sobel (sobelImg1, srcImg);
        time2 = cvGetTickCount();

        double tms = 0.001*(time2-time)/cvGetTickFrequency();

        std::cout << "Overall computation time: " << tms << "ms" << std::endl;
        std::string rname = resultName (fname, ".sobel");
        imwrite (rname, sobelImg1);   // save the resulting image
        std::cout << "Saving " << rname << std::endl;
    }
}

std::string resultName (const std::string& fname, const std::string& resultTxt)
{
    int n = fname.length();
    int idx = fname.rfind(".");
    if (idx<0) idx=n;
    return fname.substr(0,idx) + resultTxt + fname.substr(idx, n-idx);
}
