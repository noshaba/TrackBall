#include "hip/hip_runtime.h"
#include "central.h"
#include <hip/hip_runtime_api.h>
#include "sobel.cuh"


__global__ void sobelKernel (uchar* dstImg, uchar* srcImg, int rows, int cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x+y * (gridDim.x * blockDim.x);

    bool inside = in_img(x, y, rows, cols);

    int sX = inside ?
            ((-srcImg[offset-cols-1] - 2 * srcImg[offset-1] - srcImg[offset+cols-1]
            + srcImg[offset-cols+1] + 2 * srcImg[offset+1] + srcImg[offset+cols+1]) * 0.125f) : 0;

    int sY = inside ?
            ((-srcImg[offset-cols-1] - 2 * srcImg[offset-cols] - srcImg[offset-cols+1]
            + srcImg[offset+cols-1] + 2 * srcImg[offset+cols] + srcImg[offset+cols+1]) * 0.125f) : 0;

    sX = (int) floor (sX+3.5/8);
    sY = (int) floor (sY+3.5/8);

    dstImg[offset] = (sX+128) + ((sY+128)<<8);
}


void sobel (Mat_<uchar>& dstImg, const Mat_<uchar>& srcImg)
{
    int size = dstImg.rows * dstImg.cols;

    uchar *d_dstImgData;
    uchar *d_srcImgData;

    hipMalloc((void**) &d_dstImgData, size);
    hipMalloc((void**) &d_srcImgData, size);

    hipMemcpy(d_srcImgData, srcImg.data, size, hipMemcpyHostToDevice);

    dim3 threads(16,16);
    dim3 blocks(dstImg.rows/threads.x, dstImg.cols/threads.y);

    sobelKernel<<<blocks,threads>>>(d_dstImgData, d_srcImgData, dstImg.rows, dstImg.cols);

    hipDeviceSynchronize();
    hipMemcpy(dstImg.data, d_dstImgData, size, hipMemcpyDeviceToHost);

    hipFree(d_srcImgData);
    hipFree(d_dstImgData);
}
