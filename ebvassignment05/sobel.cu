#include "hip/hip_runtime.h"
#include "central.h"
#include <hip/hip_runtime_api.h>
#include "sobel.cuh"

__global__ void sobelKernel (uchar* dstImg, uchar* srcImg, int rows, int cols)
{
    //TODO: implement the sobel operator (6P)
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int step = gridDim.x * blockDim.x; 
	
	double sX = 0, sY = 0;
	
	if(in_img(x-1,y-1,rows,cols) && in_img(x+1,y+1,rows,cols)) {
		// (x,y) = step * y + x
		sX = (-srcImg[step * (y - 1) + x - 1] - 2 * srcImg[step * y + x - 1] - srcImg[step * (y + 1) + x - 1]
			 + srcImg[step * (y - 1) + x + 1] + 2 * srcImg[step * y + x + 1] + srcImg[step * (y + 1) + x + 1]) * 0.125;
			 
		sY = (-srcImg[step * (y - 1) + x - 1] - 2 * srcImg[step * (y - 1) + x] - srcImg[step * (y - 1) + x + 1]
			 + srcImg[step * (y + 1) + x - 1] + 2 * srcImg[step * (y + 1) + x] + srcImg[step * (y + 1) + x + 1]) * 0.125;
	}
	
	// save sobel length in image
	if(in_img(x,y,rows,cols))
		dstImg[step * y + x] = sqrt(sX * sX + sY * sY);
}

void sobel (Mat_<uchar>& dstImg, const Mat_<uchar>& srcImg)
{
    //TODO: implement (4P)
	
	unsigned int data_length = dstImg.rows * dstImg.cols;
	size_t size = data_length * sizeof(uchar);
	
	uchar *gpuSrcImg, *gpuDstImg;
	
	// allocate memory for the images
	CHECK_CUDA(hipMalloc((void**) &gpuSrcImg, size));
	CHECK_CUDA(hipMalloc((void**) &gpuDstImg, size));
	
	// transfer the initialized source image to the device
	CHECK_CUDA(hipMemcpy(gpuSrcImg, srcImg.data, size, hipMemcpyHostToDevice));
	
	dim3 threads(16, 16);
	dim3 blocks(round_up(dstImg.cols, threads.x), round_up(dstImg.rows, threads.y));
	
	sobelKernel<<<blocks, threads>>>(gpuDstImg, gpuSrcImg, srcImg.rows, srcImg.cols);
	
	// copy results back to the host (implies hipDeviceSynchronize())
	CHECK_CUDA(hipMemcpy(dstImg.data, gpuDstImg, size, hipMemcpyDeviceToHost));
	
	// free the memory
	CHECK_CUDA(hipFree(gpuSrcImg));
	CHECK_CUDA(hipFree(gpuDstImg));
	
	CHECK_CUDA(hipDeviceReset());
}
