#include "hip/hip_runtime.h"
#include "central.h"
#include <hip/hip_runtime_api.h>
#include "sobel.cuh"

__device__ void sobel1Px (Mat_<uchar>& dstImg, const Mat_<uchar>& srcImg, int x2, int y2)
{
	bool inside = x2 -1 >= 0 && x2 +1 < srcImg.cols && y2 -1 >= 0 && y2 +1 < srcImg.rows;

	//sobel x filter
	int sX = inside ?
		(
		(-srcImg(y2 - 1, x2 -1) - 2 * srcImg(y2,x2-1) - srcImg(y2+1,x2-1)
		+ srcImg(y2 - 1, x2 +1) + 2 * srcImg(y2,x2+1) + srcImg(y2+1,x2+1)) * 0.125f) : 0;

	//sobel y filter
	int sY = inside ?
		(
		(-srcImg(y2 - 1, x2 -1) - 2 * srcImg(y2-1,x2) - srcImg(y2-1,x2+1)
		+ srcImg(y2 + 1, x2 -1) + 2 * srcImg(y2+1,x2) + srcImg(y2+1,x2+1)) * 0.125f) : 0;

	// filtered pixel
	dstImg(y2,x2) = sobelCode(sX,sY);
}

__global__ void sobelKernel (uchar* dstImg, uchar* srcImg, int rows, int cols, int totalRows, int totalCols)
{
    //TODO: implement the sobel operator (6P)

	
	bool inside = 	(cols -1 >= 0) 		&& 
			(cols+1 < totalCols) 	&& 
			(rows -1 >= 0 && rows +1 < totalRows);
	

	//sobel x filter
	int sX = inside ?
		(
		(-srcImg[rows - 1, cols -1] - 2 * srcImg(rows,cols-1) - srcImg(rows+1,cols-1)
		+ srcImg(rows - 1, cols +1) + 2 * srcImg(rows,cols+1) + srcImg(rows+1,cols+1)) * 0.125f) : 0;

	//sobel y filter
	int sY = inside ?
		(
		(-srcImg(rows - 1, cols -1) - 2 * srcImg(rows-1,cols) - srcImg(rows-1,cols+1)
		+ srcImg(rows + 1, cols -1) + 2 * srcImg(rows+1,cols) + srcImg(rows+1,cols+1)) * 0.125f) : 0;

	// filtered pixel
	dstImg(rows,cols) = sobelCode(sX,sY);
}


void sobel (Mat_<uchar>& dstImg, const Mat_<uchar>& srcImg)
{
    //TODO: implement (4P)

	int imgSize = srcImg.rows*srcImg.cols;
	Mat_<uchar> *srcImg_d;
	Mat_<uchar> *dstImg_d;
	hipMalloc((void **) &srcImg_d,imgSize); 
	hipMalloc((void **) &dstImg_d,imgSize);

	hipMemcpy(srcImg_d, srcImg, imgSize, hipMemcpyHostToDevice);
	hipMemcpy(dstImg_d, dstImg, imgSize, hipMemcpyHostToDevice);

	assert (dstImg.size() == srcImg.size());

	//edge detection
	for (int y2 = 0; y2 < dstImg.rows;y2++)
	{
		for (int x2 = 0; x2 <dstImg.cols; x2++)
			sobelKernel <<< srcImg.rows, srcImg.cols >>> (dstImg_d,srcImg_d,y2,x2,srcImg.rows,srcImg.cols);
	}
}
